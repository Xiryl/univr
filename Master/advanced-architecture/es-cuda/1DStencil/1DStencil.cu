#include "hip/hip_runtime.h"
#include <chrono>
#include <iomanip>
#include <iostream>
#include <random>
#include "Timer.cuh"
#include "CheckError.cuh"

using namespace std;
using namespace timer;

const int RADIUS = 7;
const int N  = 10000000;
const int ITERATIONS = 10;
const int BLOCK_DIMENSION = 254;

__global__
void stencilKernel(const int* d_input, int N,int* d_output) {
    // indice univoco della thread
    int thdIx = blockIdx.x * blockDim.x + threadIdx.x;

    // limiti sx e dx
    int leftBound = RADIUS;
    int rightBound = N - RADIUS;

    if(thdIx < N && thdIx >= leftBound && thdIx < rightBound) {
        // sommo gli elementi dell range [ - RADIUS <- thIdx -> + RADIUS]
        for(int i = - RADIUS ; i <= RADIUS; i++) {
            d_output[thdIx] += d_input[thdIx + i];
        }
    }
}


int main() {
    Timer<DEVICE> TM_device_total;
    Timer<DEVICE> TM_device_core;
    Timer<HOST>   TM_host;

    float time_device_total = 0;
    float time_device_core = 0;
    float total_time_device_total = 0;
    float total_time_device_core = 0;
    float time_host = 0;
    float total_time_host = 0;
    int current_iteration = 0;

    cout << "RADIUS: " << RADIUS << endl;
    cout << "ITERATIONS: " << ITERATIONS << endl;
    cout << "N: " << N << endl;

    while(current_iteration++ < ITERATIONS) {
        cout << "EXECUTING ITERATION N. " << current_iteration << endl;

        // host allocation
        int* h_input      = new int[N];
        int* h_output_tmp = new int[N]; // <-- used for device result
        int* h_output     = new int[N](); // initilization to zero

        //cout << "Array initialization started ..." << endl;

        // host initialization
        unsigned seed = std::chrono::system_clock::now().time_since_epoch().count();
        std::default_random_engine generator(seed);
        std::uniform_int_distribution<int> distribution(1, 100);

        for (int i = 0; i < N; i++)
            h_input[i] = distribution(generator);

        //cout << "Host execution started ..." << endl;

        // host execution
        TM_host.start();

        for (int i = RADIUS; i < N - RADIUS; i++) {
            for (int j = i - RADIUS; j <= i + RADIUS; j++)
                h_output[i] += h_input[j];
        }

        TM_host.stop();
        time_host += TM_host.duration();
        //cout << "\t--> done" << endl;

        //cout << "Device allocation started ..." << endl;

        TM_device_total.start();

        // device memory allocation
        int *d_input, *d_output;
        SAFE_CALL( hipMalloc( &d_input, N * sizeof(int) ))
        SAFE_CALL( hipMalloc( &d_output, N * sizeof(int) ))

        //cout << "Memcpy CPU --> GPU started ..." << endl;
        // copy data CPU --> GPU
        SAFE_CALL( hipMemcpy( d_input, h_input, N * sizeof(int), hipMemcpyHostToDevice ))

        // set dimensions
        dim3 DimGrid(N/BLOCK_DIMENSION, 1, 1);
        if (N%BLOCK_DIMENSION) DimGrid.x++;
        dim3 DimBlock(BLOCK_DIMENSION, 1, 1);

        //cout << "Device execution started ..." << endl;
        // device execution
        TM_device_core.start();

        stencilKernel<<<DimGrid, DimBlock>>>(d_input, N, d_output);

        TM_device_core.stop();
        time_device_core += TM_device_core.duration();
        CHECK_CUDA_ERROR
        //cout << "\t--> done" << endl;

        //cout << "Memcpy CPU <-- GPU started ..." << endl;
        // copy data CPU <-- GPU
        SAFE_CALL( hipMemcpy( h_output_tmp, d_output, N * sizeof(int), hipMemcpyDeviceToHost));

        TM_device_total.stop();
        time_device_total += TM_device_total.duration();

        // check result
        for (int i = 0; i < N; i++) {
            if (h_output[i] != h_output_tmp[i]) {
                cerr << "wrong result at: " << i
                        << "\nhost:   " << h_output[i]
                        << "\ndevice: " << h_output_tmp[i] << "\n\n";
                hipDeviceReset();
                exit(EXIT_FAILURE);
            }
        }
        cout << "\t --> Computation OK\n\n";

        // host deallocation
        delete[] h_input;
        delete[] h_output;
        delete[] h_output_tmp;

        // device deallocation
        SAFE_CALL( hipFree( d_input ) )
        SAFE_CALL( hipFree( d_output ) )
    }

    float m_total_host = time_host / ITERATIONS;
    float m_total_device_total = time_device_total / ITERATIONS;
    float m_total_device_core = time_device_core / ITERATIONS;
    float speedup_core = m_total_host / m_total_device_core;
    float speedup_total = m_total_host / m_total_device_total;
    cout << "RADIUS: " << RADIUS << endl;
    cout << "ITERATIONS: " << ITERATIONS << endl;
    cout << "N: " << N << endl;
    cout << "BLOCK_DIMENSION: " << BLOCK_DIMENSION << endl;
    cout << "total host: " << time_host << endl;
    cout << "total device: " << time_device_total << endl;
    cout << "core device: " << time_device_core << endl;
    cout << "media speedup core: " << speedup_core << "x" << endl;
    cout << "media speedup total: " << speedup_total << "x" << endl;

    // reset GPU
    //hipDeviceReset();
}
