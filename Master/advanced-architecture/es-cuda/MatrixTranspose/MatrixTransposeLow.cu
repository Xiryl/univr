#include "hip/hip_runtime.h"
#include <chrono>
#include <iomanip>
#include <iostream>
#include <random>
#include "Timer.cuh"
#include "CheckError.cuh"


using namespace std;
using namespace timer;

__global__
void matrixTransposeKernel(int* d_matrix_in,
                           int        N) {
      // row and col for actual thread
      int row = threadIdx.x + (blockDim.x * blockIdx.x);
      int col = threadIdx.y + (blockDim.y * blockIdx.y);

    if ( row != col && row < col && col < N) {
        d_matrix_in[row * N + col] = d_matrix_in[row * N + col] + d_matrix_in[col * N + row]; //  a=a+b;
        d_matrix_in[col * N + row] = d_matrix_in[row * N + col] - d_matrix_in[col * N + row]; //  b=a-b;
        d_matrix_in[row * N + col] =  d_matrix_in[row * N + col] - d_matrix_in[col * N + row]; // a=a-b;
    }
}

const int N  = 1000;
const int ITERATIONS = 10;
const int BLOCK_SIZE_X = 16;
const int BLOCK_SIZE_Y = 16;

int main() {
    Timer<DEVICE> TM_device_total;
    Timer<DEVICE> TM_device_core;
    Timer<HOST>   TM_host;

    float time_device_total = 0;
    float time_device_core = 0;
    float time_host = 0;
    int current_iteration = 0;

    // -------------------------------------------------------------------------
    // HOST MEMORY ALLOCATION
    int* h_matrix_in  = new int[N * N];
    int* h_matrix_tmp = new int[N * N]; // <-- used for device result
    int* h_matrix_out = new int[N * N];

    // -------------------------------------------------------------------------
    // HOST INITILIZATION
    unsigned seed = std::chrono::system_clock::now().time_since_epoch().count();
    std::default_random_engine generator(seed);
    std::uniform_int_distribution<int> distribution(1, 100);

    for (int i = 0; i < N * N; i++)
        h_matrix_in[i] = distribution(generator);

    // -------------------------------------------------------------------------
    // HOST EXECUTIION
    TM_host.start();

    for (int i = 0; i < N ; i++) {
        for (int j = 0; j < N ; j++)
            h_matrix_out[i * N + j] = h_matrix_in[j * N + i];
    }

    TM_host.stop();
    TM_host.print("MatrixTranspose host:   ");

    // -------------------------------------------------------------------------
    // DEVICE MEMORY ALLOCATION
    int *d_matrix_in, *d_matrix_out;
    SAFE_CALL( hipMalloc( &d_matrix_in, N * N * sizeof(int) ) )
    // SAFE_CALL( hipMalloc( &d_matrix_out, N * N * sizeof(int) ) )

    // -------------------------------------------------------------------------
    // COPY DATA FROM HOST TO DEVIE
    SAFE_CALL( hipMemcpy( d_matrix_in, h_matrix_in, N * N * sizeof(int), hipMemcpyHostToDevice ))

    // -------------------------------------------------------------------------
    // DEVICE EXECUTION
    TM_device_core.start();

    dim3 DimGrid(N/BLOCK_SIZE_X, N/BLOCK_SIZE_Y, 1);
    if (N%BLOCK_SIZE_X) DimGrid.x++;
    if (N%BLOCK_SIZE_Y) DimGrid.y++;
    dim3 DimBlock(BLOCK_SIZE_X, BLOCK_SIZE_Y, 1);

    matrixTransposeKernel<<< DimGrid , DimBlock >>>(d_matrix_in, N);

    TM_device_core.stop();
    CHECK_CUDA_ERROR
    TM_device_core.print("MatrixTranspose device: ");

    std::cout << std::setprecision(1)
              << "Speedup: " << TM_host.duration() / TM_device_core.duration()
              << "x\n\n";

    // -------------------------------------------------------------------------
    // COPY DATA FROM DEVICE TO HOST
    SAFE_CALL( hipMemcpy( h_matrix_tmp, d_matrix_in, N * N * sizeof(int), hipMemcpyDeviceToHost));

    // -------------------------------------------------------------------------
    // RESULT CHECK
    for (int i = 0; i < N * N; i++) {
        if (h_matrix_out[i] != h_matrix_tmp[i]) {
            std::cerr << "wrong result at: ("
                      << (i / N) << ", " << (i % N) << ")"
                      << "\nhost:   " << h_matrix_out[i]
                      << "\ndevice: " << h_matrix_tmp[i] << "\n\n";
            hipDeviceReset();
            std::exit(EXIT_FAILURE);
        }
    }
    std::cout << "<> Correct\n\n";

    // -------------------------------------------------------------------------
    // HOST MEMORY DEALLOCATION
    delete[] h_matrix_in;
    //delete[] h_matrix_out;
    delete[] h_matrix_tmp;

    // -------------------------------------------------------------------------
    // DEVICE MEMORY DEALLOCATION
    SAFE_CALL( hipFree( d_matrix_in ) )
    // SAFE_CALL( hipFree( d_matrix_out ) )

    // -------------------------------------------------------------------------
    // hipDeviceReset();
}
