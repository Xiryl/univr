#include "hip/hip_runtime.h"
#include <chrono>
#include <iomanip>
#include <iostream>
#include <random>
#include "Timer.cuh"
#include "CheckError.cuh"

using namespace std;
using namespace timer;

const int BLOCK_SIZE_X = 16; // dato da width / tile width
const int BLOCK_SIZE_Y = 16;
const int ITERATIONS = 10;
const int N = 300;

__global__
void matrixMultiplicationKernel(const int* d_matrixA,
                                const int* d_matrixB,
                                int        N,
                                int*       d_matrixC) {
    // row and col for actual thread
    int row = threadIdx.x + (blockDim.x * blockIdx.x);
    int col = threadIdx.y + (blockDim.y * blockIdx.y);

    if (row < N && col < N) {
        // element of matrix computed
        int value = 0;

        for (int i = 0; i < N; i++) {
            value += d_matrixA[row * N + i] * d_matrixB[i * N + col];
        }
    
        d_matrixC[row * N + col] = value;
    }
}

int main() {
    Timer<DEVICE> TM_device_total;
    Timer<DEVICE> TM_device_core;
    Timer<HOST>   TM_host;

    float time_device_total = 0;
    float time_device_core = 0;
    float time_host = 0;
    int current_iteration = 0;

    // host initalization
    while(current_iteration++ < ITERATIONS) {
        cout << "EXECUTING ITERATION N. " << current_iteration << endl;

        // host allocation
        int* h_matrixA    = new int[N * N];
        int* h_matrixB    = new int[N * N];
        int* h_matrix_tmp = new int[N * N]; // <-- used for device result
        int* h_matrixC    = new int[N * N];

        unsigned seed = std::chrono::system_clock::now().time_since_epoch().count();
        std::default_random_engine generator(seed);
        std::uniform_int_distribution<int> distribution(1, 100);

        for (int i = 0; i < N * N; i++) {
            h_matrixA[i] = distribution(generator);
            h_matrixB[i] = distribution(generator);
        }

        // host execution
        TM_host.start();

        for (int i = 0; i < N; i++) {
            for (int j = 0; j < N; j++) {
                int sum = 0;
                for (int k = 0; k < N; k++)
                    sum += h_matrixA[i * N + k] * h_matrixB[k * N + j];
                h_matrixC[i * N + j] = sum;
            }
        }

        TM_host.stop();
        time_host += TM_host.duration();

        TM_device_total.start();

        // device allocation
        int *d_matrixA, *d_matrixB, *d_matrixC;
        SAFE_CALL( hipMalloc(  &d_matrixA, N * N * sizeof(int) ) )
        SAFE_CALL( hipMalloc(  &d_matrixB, N * N * sizeof(int) ) )
        SAFE_CALL( hipMalloc(  &d_matrixC, N * N * sizeof(int) ) )

        // copy data CPU -> GPU
        SAFE_CALL( hipMemcpy( d_matrixA, h_matrixA, N * N * sizeof(int), hipMemcpyHostToDevice ))
        SAFE_CALL( hipMemcpy( d_matrixB, h_matrixB, N * N * sizeof(int), hipMemcpyHostToDevice ))

        dim3 DimGrid(N/BLOCK_SIZE_X, N/BLOCK_SIZE_Y, 1);
        if (N%BLOCK_SIZE_X) DimGrid.x++;
        if (N%BLOCK_SIZE_Y) DimGrid.y++;
        dim3 DimBlock(BLOCK_SIZE_X, BLOCK_SIZE_Y, 1);

        // device execution
        TM_device_core.start();

        matrixMultiplicationKernel<<< DimGrid, DimBlock >>> (d_matrixA, d_matrixB, N, d_matrixC);

        TM_device_core.stop();
        time_device_core += TM_device_core.duration();
        CHECK_CUDA_ERROR

        // copy data CPU <-- GPU
        SAFE_CALL( hipMemcpy( h_matrix_tmp, d_matrixC, N * N * sizeof(int), hipMemcpyDeviceToHost));

        TM_device_total.stop();
        time_device_total += TM_device_total.duration();

        // check res
        for (int i = 0; i < N * N; i++) {
            if (h_matrixC[i] != h_matrix_tmp[i]) {
                std::cerr << "wrong result at: ("
                        << (i / N) << ", " << (i % N) << ")"
                        << "\nhost:   " << h_matrixC[i]
                        << "\ndevice: " << h_matrix_tmp[i] << "\n\n";
                hipDeviceReset();
                std::exit(EXIT_FAILURE);
            }
        }
        cout << "\t --> Computation OK\n\n";

        // host deallocation
        delete[] h_matrixA;
        delete[] h_matrixB;
        delete[] h_matrixC;
        delete[] h_matrix_tmp;

        // device deallocation
        
        SAFE_CALL( hipFree( d_matrixA ) )
        SAFE_CALL( hipFree( d_matrixB ) )
        SAFE_CALL( hipFree( d_matrixC ) )
    }

    float m_total_host = time_host / ITERATIONS;
    float m_total_device_total = time_device_total / ITERATIONS;
    float m_total_device_core = time_device_core / ITERATIONS;
    float speedup_core = m_total_host / m_total_device_core;
    float speedup_total = m_total_host / m_total_device_total;
    cout << "BLOCK_SIZE: " << BLOCK_SIZE_X << endl;
    cout << "ITERATIONS: " << ITERATIONS << endl;
    cout << "N: " << N << endl;
    cout << "total host: " << time_host << endl;
    cout << "total device: " << time_device_total << endl;
    cout << "core device: " << time_device_core << endl;
    cout << "media speedup core: " << speedup_core << "x" << endl;
    cout << "media speedup total: " << speedup_total << "x" << endl;

    // reset
    // hipDeviceReset();
}
