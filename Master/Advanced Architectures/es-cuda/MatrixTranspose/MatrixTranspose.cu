#include "hip/hip_runtime.h"
#include <chrono>
#include <iomanip>
#include <iostream>
#include <random>
#include "Timer.cuh"
#include "CheckError.cuh"

using namespace std;
using namespace timer;

const int N  = 1000;
const int ITERATIONS = 10;
const int BLOCK_SIZE_X = 16;
const int BLOCK_SIZE_Y = 16;

__global__
void matrixTransposeKernel(const int* d_matrix_in,
                           int        N,
                           int*       d_matrix_out) {
      // row and col for actual thread
      int row = threadIdx.x + (blockDim.x * blockIdx.x);
      int col = threadIdx.y + (blockDim.y * blockIdx.y);
  
      if (row < N && col < N) {
          d_matrix_out[row * N + col] = d_matrix_in[col * N + row];
      }
}

int main() {
    Timer<DEVICE> TM_device_total;
    Timer<DEVICE> TM_device_core;
    Timer<HOST>   TM_host;

    float time_device_total = 0;
    float time_device_core = 0;
    float time_host = 0;
    int current_iteration = 0;

    while(current_iteration++ < ITERATIONS) {
        cout << "EXECUTING ITERATION N. " << current_iteration << endl;

        // -------------------------------------------------------------------------
        // HOST MEMORY ALLOCATION
        int* h_matrix_in  = new int[N * N];
        int* h_matrix_tmp = new int[N * N]; // <-- used for device result
        int* h_matrix_out = new int[N * N];

        // -------------------------------------------------------------------------
        // HOST INITILIZATION
        unsigned seed = std::chrono::system_clock::now().time_since_epoch().count();
        std::default_random_engine generator(seed);
        std::uniform_int_distribution<int> distribution(1, 100);

        for (int i = 0; i < N * N; i++)
            h_matrix_in[i] = distribution(generator);

        // -------------------------------------------------------------------------
        // HOST EXECUTIION
        TM_host.start();

        for (int i = 0; i < N ; i++) {
            for (int j = 0; j < N ; j++)
                h_matrix_out[i * N + j] = h_matrix_in[j * N + i];
        }

        TM_host.stop();
        time_host += TM_host.duration();

        TM_device_total.start();

        // -------------------------------------------------------------------------
        // DEVICE MEMORY ALLOCATION
        int *d_matrix_in, *d_matrix_out;
        SAFE_CALL( hipMalloc( &d_matrix_in, N * N * sizeof(int) ) )
        SAFE_CALL( hipMalloc( &d_matrix_out, N * N * sizeof(int) ) )

        // -------------------------------------------------------------------------
        // COPY DATA FROM HOST TO DEVIE
        SAFE_CALL( hipMemcpy( d_matrix_in, h_matrix_in, N * N * sizeof(int), hipMemcpyHostToDevice ))

        // -------------------------------------------------------------------------
        // DEVICE EXECUTION
        TM_device_core.start();

        dim3 DimGrid(N/BLOCK_SIZE_X, N/BLOCK_SIZE_Y, 1);
        if (N%BLOCK_SIZE_X) DimGrid.x++;
        if (N%BLOCK_SIZE_Y) DimGrid.y++;
        dim3 DimBlock(BLOCK_SIZE_X, BLOCK_SIZE_Y, 1);

        matrixTransposeKernel<<< DimGrid , DimBlock >>>(d_matrix_in, N, d_matrix_out);

        CHECK_CUDA_ERROR
        TM_device_core.stop();
        time_device_core += TM_device_core.duration();


        // -------------------------------------------------------------------------
        // COPY DATA FROM DEVICE TO HOST
        SAFE_CALL( hipMemcpy( h_matrix_tmp, d_matrix_out, N * N * sizeof(int), hipMemcpyDeviceToHost));

        TM_device_total.stop();
        time_device_total += TM_device_total.duration();

        // -------------------------------------------------------------------------
        // RESULT CHECK
        for (int i = 0; i < N * N; i++) {
            if (h_matrix_out[i] != h_matrix_tmp[i]) {
                std::cerr << "wrong result at: ("
                        << (i / N) << ", " << (i % N) << ")"
                        << "\nhost:   " << h_matrix_out[i]
                        << "\ndevice: " << h_matrix_tmp[i] << "\n\n";
                hipDeviceReset();
                std::exit(EXIT_FAILURE);
            }
        }
        std::cout << "<> Correct\n\n";

        // -------------------------------------------------------------------------
        // HOST MEMORY DEALLOCATION
        delete[] h_matrix_in;
        delete[] h_matrix_out;
        delete[] h_matrix_tmp;

        // -------------------------------------------------------------------------
        // DEVICE MEMORY DEALLOCATION
        SAFE_CALL( hipFree( d_matrix_in ) )
        SAFE_CALL( hipFree( d_matrix_out ) )
    }

    float m_total_host = time_host / ITERATIONS;
    float m_total_device_total = time_device_total / ITERATIONS;
    float m_total_device_core = time_device_core / ITERATIONS;
    float speedup_core = m_total_host / m_total_device_core;
    float speedup_total = m_total_host / m_total_device_total;
    cout << "BLOCK_SIZE: " << BLOCK_SIZE_X << endl;
    cout << "ITERATIONS: " << ITERATIONS << endl;
    cout << "N: " << N << endl;
    cout << "total host: " << time_host << endl;
    cout << "total device: " << time_device_total << endl;
    cout << "core device: " << time_device_core << endl;
    cout << "media speedup core: " << speedup_core << "x" << endl;
    cout << "media speedup total: " << speedup_total << "x" << endl;

    // -------------------------------------------------------------------------
    // hipDeviceReset();
}
